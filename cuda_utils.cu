#include <hip/hip_runtime.h>

#include "cuda_utils.h"

int getCUDADeviceCount() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    return deviceCount;
}
